#include "hip/hip_runtime.h"
#include "questions/helpers/array_functions.h"

template <typename data_type>
__global__ void comparison_kernel (const int n, const data_type *in_1, const data_type *in_2, int *result)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n)
    if (in_1[i] != in_2[i])
      *result = 1;
}

template <typename data_type>
bool is_equal (const int n, const data_type *in_1, const data_type *in_2)
{
  int *device_flag {};
  hipMalloc (&device_flag, sizeof (int));
  hipMemset (device_flag, 0, sizeof (int));

  const int block_size = 128;
  const int grid_size = (n + block_size - 1) / block_size;

  comparison_kernel<<<grid_size, block_size>>> (n, in_1, in_2, device_flag);

  int host_flag {};
  hipMemcpy (&host_flag, device_flag, sizeof (int), hipMemcpyDeviceToHost);

  hipFree (device_flag);

  return host_flag == 0;
}

#define IS_EQUAL_INSTANTIATION(TYPE) \
template bool is_equal<TYPE> (const int n, const TYPE *in_1, const TYPE *in_2)

IS_EQUAL_INSTANTIATION(float);
