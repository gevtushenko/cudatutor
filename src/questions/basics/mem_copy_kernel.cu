#include "hip/hip_runtime.h"
//

#include "questions/basics/mem_copy.h"

__global__ void reference_kernel (
    const int n,
    const float *in,
    float *out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n)
    out[i] = in[i];
}

void mem_copy_kernel_wrapper (const int n, const float *in, float *out)
{
  const int block_size = 128;
  const int grid_size = (n + block_size - 1) / block_size;

  reference_kernel<<<grid_size, block_size>>> (n, in, out);
}
