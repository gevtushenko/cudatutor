#include "hip/hip_runtime.h"
//

#include "questions/basics/mem_copy_01.h"

__global__ void mem_copy_01_reference_kernel (
    const float *in,
    float *out)
{
  out[0] = in[0];
}

void mem_copy_01_kernel_wrapper (const int n, const float *in, float *out)
{
  mem_copy_01_reference_kernel<<<1, 1>>> (in, out);
}

__global__ void mem_copy_02_reference_kernel (
    const int n,
    const float *in,
    float *out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n)
    out[i] = in[i];
}

void mem_copy_02_kernel_wrapper (const int n, const float *in, float *out)
{
  const int block_size = 128;
  const int grid_size = (n + block_size - 1) / block_size;

  mem_copy_02_reference_kernel<<<grid_size, block_size>>> (n, in, out);
}
