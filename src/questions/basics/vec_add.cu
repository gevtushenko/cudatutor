#include "hip/hip_runtime.h"
//

#include "questions/basics/vec_add.h"

#include "answer.h"

const char *vec_add_question_t::get_question_content () const
{
  return R"(
#include <boost/config.hpp> // for BOOST_SYMBOL_EXPORT

extern "C" BOOST_SYMBOL_EXPORT int run ()
{
return 42;
}
)";
}

__global__ void reference_kernel (
    const int n,
    const float *in_1,
    const float *in_2,
    float *out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n)
    out[i] = in_1[i] + in_2[i];
}

bool vec_add_question_t::check_answer_implementation () const
{
  auto run = load_answer<int()> ("answer.so", "run");
  return run () == 42;
}
