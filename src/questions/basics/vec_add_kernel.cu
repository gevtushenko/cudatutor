
#include <hip/hip_runtime.h>
//

__global__ void vec_add_reference_kernel (
    const int n,
    const float *in_1,
    const float *in_2,
    float *out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n)
    out[i] = in_1[i] + in_2[i];
}

void vec_add_reference_kernel_wrapper (
    const int n,
    const float *in_1,
    const float *in_2,
    float *out)
{
  const int block_size = 128;
  const int grid_size = (n + block_size - 1) / block_size;

  vec_add_reference_kernel<<<block_size, grid_size>>> (n, in_1, in_2, out);
}
